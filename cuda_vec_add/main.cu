#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void vecAdd(double *a, double *b, double *c, int n)
{

    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id < n) //EVitar salir de rangos
        c[id] = a[id] + b[id];
}

int main( int argc, char* argv[] )
{
    int n = 100000; //Tamaño vector

    //Entradas host
    double *h_a;
    double *h_b;
    //Salidas host
    double *h_c;

    // Entrada vector
    double *d_a;
    double *d_b;
    //Salida vector
    double *d_c;

    // Allocate memory for each vector on host
    h_a = (double*)malloc(n*sizeof(double));
    h_b = (double*)malloc(n*sizeof(double));
    h_c = (double*)malloc(n*sizeof(double));

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, n*sizeof(double));
    hipMalloc(&d_b, n*sizeof(double));
    hipMalloc(&d_c, n*sizeof(double));

    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = rand();
        h_b[i] = rand();
    }

    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize = 1024;
    int gridSize;

    gridSize = (int)ceil((float)n/blockSize);

    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    //Sumar resultado del vector
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);

    //Liberar Memoria
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
