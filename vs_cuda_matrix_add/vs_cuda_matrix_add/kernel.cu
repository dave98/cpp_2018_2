
#include "hip/hip_runtime.h"

#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;

int SIZE = 50;
const float BLOCK_WIDTH = 8;

void add_matrix(int* , int* , int* , int, char); //Just work with same sizes matrices.
__global__ void add_matrix_kernel(int*, int*, int*, int);
__global__ void add_matrix_row_kernel(int*, int*, int*, int);
__global__ void add_matrix_col_kernel(int*, int*, int*, int);

void mul_matrix_vec(int*, int*, int*, int);
__global__ void mul_matrix_vec_kernel(int*, int*, int*, int);

//void BlockTranspose(int*, int*, int, int);
//__global__ void BlockTransposeKernel(int*, int, int);

void print_matrix(int*, int, int);
void print_vec(int*, int);



int main()
{
	
	int* m_a = new int[SIZE * SIZE];
	int* m_b = new int[SIZE * SIZE];
	int* m_c = new int[SIZE * SIZE];
	int* m_c2 = new int[SIZE * SIZE];
	int* m_c3 = new int[SIZE * SIZE];

	int* vec1 = new int[SIZE];
	int* vec2 = new int[SIZE];

	for (int i = 0; i < SIZE; i++) {
		for (int j = 0; j < SIZE; j++) {
			int pos = (i * SIZE) + j;
			m_a[pos] = i + j;
			m_b[pos] = i * j;
			m_c[pos] = 0;
			m_c2[pos] = 0;
			m_c3[pos] = 0;
		}
	}

	for (int j = 0; j < SIZE; j++) {
		vec1[j] = j;
		vec2[j] = 0;
	}

	/*
	cout << "Vector" << endl;
	print_vec(vec1, SIZE); cout << endl;
	cout << "Matriz" << endl;
	print_matrix(m_a, SIZE, SIZE); cout << endl;
	cout << "Resultado" << endl;
	mul_matrix_vec(m_a, vec1, vec2, SIZE);
	
	print_vec(vec2, SIZE); 
	*/

	//print_matrix(m_a, SIZE, SIZE); cout << endl;
	//print_matrix(m_b, SIZE, SIZE); cout << endl;

	
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	//BlockTranspose(m_a, m_c, SIZE, SIZE);
	add_matrix(m_a, m_b, m_c, SIZE, 'n');
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	//print_matrix(m_c, SIZE, SIZE);
	auto duration1 = duration_cast<microseconds>(t2 - t1).count();

	t1 = high_resolution_clock::now();
	add_matrix(m_a, m_b, m_c2, SIZE, 'r');
	t2 = high_resolution_clock::now();
	//print_matrix(m_c, SIZE, SIZE);
	auto duration2 = duration_cast<microseconds>(t2 - t1).count();

	t1 = high_resolution_clock::now();
	add_matrix(m_a, m_b, m_c3, SIZE, 'c');
	t2 = high_resolution_clock::now();
	//print_matrix(m_c, SIZE, SIZE);
	auto duration3 = duration_cast<microseconds>(t2 - t1).count();

	cout << "Normal 1: " << duration1 << endl;
	cout << "Filas 2: " << duration2 << endl;
	cout << "Columnas 3: " << duration3 << endl;
	

	return 0;	
}


void add_matrix(int* a, int* b, int* c, int size, char ans) {
	int total_size = size * size * sizeof(int);

	int* d_a;
	int* d_b;
	int* d_c;

	hipMalloc( (void**) &d_a, total_size);
	hipMalloc( (void**) &d_b, total_size);
	hipMalloc( (void**) &d_c, total_size);

	hipMemcpy(d_a, a, total_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, total_size, hipMemcpyHostToDevice);

	float block = 16;
	dim3 grid_size(ceil(size/block), ceil(size/block), 1);
	dim3 block_size(block, block, 1);

	switch (ans)
	{
	case 'n':
		add_matrix_kernel <<< grid_size, block_size >>> (d_a, d_b, d_c, size);
		break;
	case 'r':
		add_matrix_row_kernel <<< grid_size, block_size >>> (d_a, d_b, d_c, size);
		break;
	case 'c':
		add_matrix_col_kernel <<< grid_size, block_size >>> (d_a, d_b, d_c, size);
		break;
	default:
		break;
	}
	hipMemcpy(c, d_c, total_size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

//normal addition;
__global__ void add_matrix_kernel(int* a, int* b, int* c, int size) {
	int gpu_columna = (blockDim.x * blockIdx.x) + threadIdx.x;
	int gpu_fila = (blockDim.y * blockIdx.y) + threadIdx.y;

	if (gpu_columna < size  && gpu_fila < size) {
		int pos = gpu_fila * size + gpu_columna;
		c[pos] = a[pos] + b[pos];
	}
}

//row addition;
__global__ void add_matrix_col_kernel(int* a, int* b, int* c, int size) {
	int gpu_columna = (blockDim.x * blockIdx.x) + threadIdx.x;
	int gpu_fila = (blockDim.y * blockIdx.y) + threadIdx.y;

	if (gpu_fila == 0){
		for (int i = 0; i < size; i++) {
			int pos = (i *  size + gpu_columna);
			c[pos] = a[pos] + b[pos];
		}
	}
}

//col addition;
__global__ void add_matrix_row_kernel(int* a, int* b, int* c, int size) {
	int gpu_columna = (blockDim.x * blockIdx.x) + threadIdx.x;
	int gpu_fila = (blockDim.y * blockIdx.y) + threadIdx.y;

	if (gpu_columna == 0) {
		for (int i = 0; i < size; i++) {
			int pos = (gpu_fila *  size + i);
			c[pos] = a[pos] + b[pos];
		}
	}
}

/*
void BlockTranspose(int* h_mat, int* out_mat, int columnas, int filas) {
	int* d_mat;
	int m_size = columnas * filas * sizeof(int);

	cudaMalloc((void**) &d_mat, m_size);
	cudaMemcpy(d_mat, h_mat, m_size, cudaMemcpyHostToDevice);

	dim3 gridDim(ceil(columnas/BLOCK_WIDTH), ceil(filas/BLOCK_WIDTH), 1);
	dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	BlockTransposeKernel<<< gridDim, blockDim >>>(d_mat, columnas, filas);

	cudaMemcpy(out_mat, d_mat, m_size, cudaMemcpyDeviceToHost);
	
	cudaFree(d_mat);
}
*/


void mul_matrix_vec(int* h_m, int* h_vec_1, int* h_vec_2, int size) {
	int* d_m;
	int* d_vec_1;
	int* d_vec_2;

	int size_mat = size * size * sizeof(int);
	int size_vec = size * sizeof(int);
	
	hipMalloc((void**) &d_m, size_mat);
	hipMalloc((void**) &d_vec_1, size_vec);
	hipMalloc((void**) &d_vec_2, size_vec);

	hipMemcpy(d_m, h_m, size_mat, hipMemcpyHostToDevice);
	hipMemcpy(d_vec_1, h_vec_1, size_vec, hipMemcpyHostToDevice);

	dim3 gridDim(ceil(size / BLOCK_WIDTH), ceil(size / BLOCK_WIDTH), 1);
	dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	
	mul_matrix_vec_kernel <<< gridDim, blockDim >>> (d_m, d_vec_1, d_vec_2, size);

	hipMemcpy(h_vec_2, d_vec_2, size_vec, hipMemcpyDeviceToHost);
	hipFree(d_m);
	hipFree(d_vec_1);
	hipFree(d_vec_2);
}


//execution order vect(x_1, y_1) x mat(x_2, y_2); y_1 == x_2
__global__ void mul_matrix_vec_kernel(int* h_m, int* h_vec_1, int* h_vec_2, int size) {
	int gpu_columna = (blockIdx.x * blockDim.x) + threadIdx.x;
	int gpu_fila = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	if (gpu_fila == 0) {
		h_vec_2[gpu_columna] = 0; //Index for resultant matrix
		for (int i = 0; i < size; i++) { // To descend in matrix
			h_vec_2[gpu_columna] += h_m[i*size + gpu_columna] * h_vec_1[i];
		}
	}
}



void print_matrix(int* a, int s_1, int s_2) {
	for (int i = 0; i < s_1; i++) {
		for (int j = 0; j < s_2; j++) {
			cout << a[i*s_2 + j] << "  ";
		}
		cout << endl;
	}
}

void print_vec(int* a, int s_1) {
	for (int i = 0; i < s_1; i++) {
		cout << a[i] << "  ";
	}
}
