#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

__host__ __device__ const int TILE_WIDTH = 2;

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width);

int main()
{

    return 0;
}

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width) {
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = blockIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	float Pvalue = 0;
	for (int ph = 0; ph < Width / TILE_WIDTH; ++ph) {
		Mds[ty][tx] = d_M[Row*Width + ph * TILE_WIDTH + tx];
		Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty) * Width + Col];
	}
	__syncthreads();

	for (int k = 0; k < TILE_WIDTH; ++k) {
		Pvalue += Mds[ty][k] * Nds[tx]
	}
}