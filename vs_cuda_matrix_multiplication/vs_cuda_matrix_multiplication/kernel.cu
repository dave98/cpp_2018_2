
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "matrix.h"
#include <chrono>


using namespace std;

void print_matrix(int*, int, int);

int main() {
	int width = 32;
	float block = 32;

	unsigned ntotal = width * width;

	int *h_m = new int[ntotal];
	int *h_n = new int[ntotal];
	int *h_p = new int[ntotal];
	int *h_r = new int[ntotal];

	unsigned i, j;
	for (i = 0; i < width; i++) {
		for (j = 0; j < width; j++) {
			h_m[(i*width) + j] = j;
			h_n[(i*width) + j] = j;
			h_p[(i*width) + j] = 0;
			h_r[(i*width) + j] = 0;
		}
	}

	cout << "Matriz 1:" << endl;
	print_matrix(h_m, width, width); cout << endl;
	cout << "Matriz 2:" << endl;
	print_matrix(h_n, width, width); cout << endl;

	//high_resolution_clock::time_point t1 = high_resolution_clock::now();
	cout << "Multiplicacion Normal" << endl;
	square_matrix_mult(h_m, h_n, h_p, width, block, 'n');
	//high_resolution_clock::time_point t2 = high_resolution_clock::now();
	//auto duration1 = duration_cast<microseconds>(t2 - t1).count();
	print_matrix(h_p, width, width); cout << endl;

	cout << "Multiplicacion Tiles" << endl;
	square_matrix_mult(h_m, h_n, h_r, width, block, 't');
	print_matrix(h_r, width, width); cout << endl;

	delete h_m;
	delete h_n;
	delete h_p;
	delete h_r;

	return 0;
}

void print_matrix(int* m_x, int fila, int columna) {
	for (int i = 0; i < fila; i++) {
		for (int j = 0; j < columna; j++) {
			cout << m_x[i*columna + j] << " ";
		}
		cout << endl;
	}
}

