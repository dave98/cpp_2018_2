#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <stdio.h>
#include <iostream>

using namespace std;
using namespace cv;

class OpenCVUnit
{
public:
	float r;
	float g;
	float b;
	__host__ __device__ OpenCVUnit(float r_, float g_, float b_) {
		this->r = r_;
		this->g = g_;
		this->b = b_;
	}

	__host__ __device__ OpenCVUnit() {
		this->r = 0;
		this->g = 0;
		this->b = 0;
	}
	
	__host__ __device__ void set_r(float r_) { this->r = r_; }
	__host__ __device__ void set_g(float g_) { this->g = g_; }
	__host__ __device__ void set_b(float b_) { this->b = b_; }

	__host__ __device__ float get_r() { return this->r; }
	__host__ __device__ float get_g() { return this->g; }
	__host__ __device__ float get_b() { return this->b; }

};


void change_to_gray(OpenCVUnit* , float* , unsigned int , unsigned int );

__global__ void change_to_gray_kernel(OpenCVUnit*, float*, unsigned int, unsigned int);

int main()
{

	Mat image_in = imread("lena.jpg", IMREAD_COLOR);
	if (image_in.empty()) {
		cout << "No se pudo leer la imagen" << endl;
		return 0;
	}

	namedWindow("Entrada", WINDOW_NORMAL);
	imshow("Entrada", image_in);

	unsigned int columnas = image_in.cols;
	unsigned int filas = image_in.rows;

	cout << "Columnas: " << columnas << " -- " << "Filas: " << filas << endl;

	OpenCVUnit* pixels_in = new OpenCVUnit[columnas * filas]; //Pool for pixels from image
	float* pixel_out = new float[columnas *  filas];

	for (int i = 0; i < filas; i++) { //Copiar datos dentro de matriz de pixeles
		for (int j = 0; j < columnas; j++) {
			//cout << "[ " << (float)image_in.at<Vec3b>(i, j)[0] << " - " << (float)image_in.at<Vec3b>(i, j)[1] << " - " << (float)image_in.at<Vec3b>(i, j)[2] <<" ]";

			pixels_in[i*columnas + j].set_r((float)image_in.at<Vec3b>(i, j)[0]);
			pixels_in[i*columnas + j].set_g((float)image_in.at<Vec3b>(i, j)[1]);
			pixels_in[i*columnas + j].set_b((float)image_in.at<Vec3b>(i, j)[2]);
		}
		//cout << "------------" << endl;
	}

	/*
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			cout << "[ " << pixels_in->get_r() << " " << pixels_in->get_g() << " " << pixels_in->get_b() << " ]";
		}
		cout << endl;
	}
	*/

	//Tratamiento de la imagen 
	//change_to_gray(pixels_in, pixel_out, columnas, filas);
	
	int size_of_image_in = columnas * filas * sizeof(OpenCVUnit);
	int size_of_image_out = columnas * filas * sizeof(float);

	OpenCVUnit* d_pixels_in;
	float* d_pixel_out;
	
	hipMalloc((void** ) &d_pixels_in, size_of_image_in);
	hipMalloc((void** ) &d_pixel_out, size_of_image_out);
	hipMemcpy(d_pixels_in, pixels_in, size_of_image_in, hipMemcpyHostToDevice);


	int block = 16;
	dim3 size_grid = (ceil(columnas / block), ceil(filas / block), 1);
	dim3 size_block = (block, block, 1);

	change_to_gray_kernel <<< size_grid, size_block >>> (d_pixels_in, d_pixel_out, columnas , filas );
	

	hipMemcpy(pixel_out, d_pixel_out, size_of_image_out, hipMemcpyDeviceToHost);

	hipFree(d_pixels_in);
	hipFree(d_pixel_out);
	
/////

	Mat image_out(filas, columnas, CV_8UC3, Scalar(0, 0, 0));
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			//cout << "[ " << pixel_out[i*columnas + j] << " ]";
			image_out.at<Vec3b>(i, j)[0] = pixel_out[i*columnas + j];
			image_out.at<Vec3b>(i, j)[1] = pixel_out[i*columnas + j];
			image_out.at<Vec3b>(i, j)[2] = pixel_out[i*columnas + j];
		}
		//cout << endl;
	}

	namedWindow("Salida", WINDOW_NORMAL);
	imshow("Salida", image_out);

	waitKey(0);
	destroyWindow("Entrada");
	destroyWindow("Salida");

	delete pixels_in;
	delete pixel_out;
	return 0;
}

//Configuration for CUDA deployment
void change_to_gray(OpenCVUnit* h_pixels_in, float* h_out, unsigned int h_columnas, unsigned int h_filas) {
	OpenCVUnit* d_pixels_in;
	float* d_out;

	int size_of_pixel_max = h_columnas * h_filas * sizeof(OpenCVUnit);
	int size_of_float_max = h_columnas * h_filas * sizeof(float);


	hipMalloc((void** ) &d_pixels_in, size_of_pixel_max);
	hipMalloc((void**) &d_out, size_of_float_max);
	hipMemcpy(d_pixels_in, h_pixels_in, size_of_pixel_max, hipMemcpyHostToDevice);

	int block = 16;
	dim3 size_grid = (ceil(h_columnas / block), ceil(h_filas / block), 1);
	dim3 size_block = (block, block, 1);

	change_to_gray_kernel<<< size_grid, size_block >>>(d_pixels_in, d_out, h_columnas, h_filas);

	hipMemcpy(h_out, d_out, size_of_float_max, hipMemcpyDeviceToHost);

	hipFree(d_pixels_in);
	hipFree(d_out);
	
}

//CUDA code
__global__ void change_to_gray_kernel(OpenCVUnit* image, float* gray, unsigned int columnas, unsigned int filas) {
	unsigned int gpu_columma = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int gpu_fila = threadIdx.y + (blockIdx.y * blockDim.y);

	if (gpu_columma < columnas && gpu_fila < filas) {
		int place = gpu_fila * columnas + gpu_columma;
		gray[place] = (image[place].get_r() * 0.21 /*0.299*/) + (image[place].get_g() * 0.72/*0.587*/) + (image[place].get_b() * 0.07/*0.114*/);
	}
}




