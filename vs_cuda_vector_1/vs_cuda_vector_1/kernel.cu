#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <stdio.h>
#include <iostream>

#define CUDA_CALLABLE_MEMBER __host__ __device__
#define BLUR_SIZE = 3

using namespace std;
using namespace cv;

class OpenCVUnit
{
public:
	float r;
	float g;
	float b;
	
	CUDA_CALLABLE_MEMBER OpenCVUnit(float r_, float g_, float b_) {
		this->r = r_;
		this->g = g_;
		this->b = b_;
	}

	CUDA_CALLABLE_MEMBER OpenCVUnit() {
		this->r = 0;
		this->g = 0;
		this->b = 0;
	}

	CUDA_CALLABLE_MEMBER void set_r(float r_) { this->r = r_; }
	CUDA_CALLABLE_MEMBER void set_g(float g_) { this->g = g_; }
	CUDA_CALLABLE_MEMBER void set_b(float b_) { this->b = b_; }

	CUDA_CALLABLE_MEMBER float get_r() { return this->r; }
	CUDA_CALLABLE_MEMBER float get_g() { return this->g; }
	CUDA_CALLABLE_MEMBER float get_b() { return this->b; }
};


void change_to_gray(OpenCVUnit* , float* , unsigned int , unsigned int );
void change_to_blur(OpenCVUnit*, OpenCVUnit*, unsigned int, unsigned int);

__global__ void change_to_gray_kernel(OpenCVUnit*, float*, unsigned int, unsigned int);
__global__ void change_to_blur_kernel(OpenCVUnit*, OpenCVUnit*, unsigned int, unsigned int);

void print_matrix(OpenCVUnit*, int, int);
void print_matrix(float*, int, int);
void set_matrix_zero(float*, int, int);


//Main 
int main()
{

	Mat image_in = imread("zombie_1.png", IMREAD_COLOR);
	if (image_in.empty()) {
		cout << "No se pudo leer la imagen" << endl;
		return 0;
	}

	unsigned int columnas = image_in.cols;
	unsigned int filas = image_in.rows;

	cout << "Columnas: " << columnas << " -- " << "Filas: " << filas << endl;

	OpenCVUnit* pixels_in = new OpenCVUnit[columnas * filas]; //Pool for pixels from image
	
	float* pixel_out = new float[columnas *  filas]; //For gray image
	OpenCVUnit* blur_image = new OpenCVUnit[columnas * filas]; //For Blur image;


	for (int i = 0; i < filas; i++) { //Copiar datos dentro de matriz de pixeles
		for (int j = 0; j < columnas; j++) {
			pixels_in[i*columnas + j].set_r((float)image_in.at<Vec3b>(i, j)[0]);
			pixels_in[i*columnas + j].set_g((float)image_in.at<Vec3b>(i, j)[1]);
			pixels_in[i*columnas + j].set_b((float)image_in.at<Vec3b>(i, j)[2]);
		}
	}

	//Tratamiento de la imagen 
	change_to_gray(pixels_in, pixel_out, columnas, filas); ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	change_to_blur(pixels_in, blur_image, columnas, filas);
	
	Mat image_out(filas, columnas, CV_8UC3); //For Gray
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			image_out.at<Vec3b>(i, j)[0] = pixel_out[i*columnas + j];
			image_out.at<Vec3b>(i, j)[1] = pixel_out[i*columnas + j];
			image_out.at<Vec3b>(i, j)[2] = pixel_out[i*columnas + j];
		}
	}

	Mat image_blur(filas, columnas, CV_8UC3); //For Gray
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			image_blur.at<Vec3b>(i, j)[0] = blur_image[i*columnas + j].get_r();
			image_blur.at<Vec3b>(i, j)[1] = blur_image[i*columnas + j].get_g();
			image_blur.at<Vec3b>(i, j)[2] = blur_image[i*columnas + j].get_b();
		}
	}

	namedWindow("Entrada", WINDOW_NORMAL);
	imshow("Entrada", image_in);

	namedWindow("Gray", WINDOW_NORMAL);
	imshow("Gray", image_out);


	namedWindow("Blur", WINDOW_NORMAL);
	imshow("Blur", image_blur);

	waitKey(0);
	destroyWindow("Entrada");
	destroyWindow("Gray");
	destroyWindow("Blur");

	delete pixels_in;
	delete pixel_out;
	return 0;
}

//Configuration for CUDA deployment
void change_to_gray(OpenCVUnit* h_pixels_in, float* h_out, unsigned int h_columnas, unsigned int h_filas) {
	OpenCVUnit* d_pixels_in;
	float* d_out;

	//Imprimiendo matriz//HERE
	//print_matrix(h_pixels_in, h_filas, h_columnas);
	//print_matrix(h_out, h_filas, h_columnas);

	int size_of_pixel_max = h_columnas * h_filas * sizeof(OpenCVUnit);
	int size_of_float_max = h_columnas * h_filas * sizeof(float);

	hipMalloc((void** ) &d_pixels_in, size_of_pixel_max);
	hipMalloc((void**) &d_out, size_of_float_max);
	hipMemcpy(d_pixels_in, h_pixels_in, size_of_pixel_max, hipMemcpyHostToDevice);

	//print_matrix(d_pixels_in, h_filas, h_columnas);

	float block = 32;
	dim3 size_grid(ceil(h_columnas / block), ceil(h_filas / block), 1);
	dim3 size_block(block, block, 1);				
	
	cout << "Dimensiones de Grid: [" << size_grid.x << " - " << size_grid.y << " - " << size_grid.z << "]  Size Block: [" << size_block.x << " - " << size_block.y << " - " << size_block.z << "] " << endl;

	change_to_gray_kernel<<< size_grid, size_block >>>(d_pixels_in, d_out, h_columnas, h_filas);

	hipMemcpy(h_out, d_out, size_of_float_max, hipMemcpyDeviceToHost);

	//print_matrix(h_out, h_filas, h_columnas);
	hipFree(d_pixels_in);
	hipFree(d_out);
}

void change_to_blur(OpenCVUnit* img_in, OpenCVUnit* img_out, unsigned int columnas, unsigned int filas) {
	unsigned img_size = columnas * filas * sizeof(OpenCVUnit);

	OpenCVUnit* d_img_in;
	OpenCVUnit* d_img_out;

	hipMalloc((void**)&d_img_in, img_size);
	hipMalloc((void**)&d_img_out, img_size);

	hipMemcpy(d_img_in, img_in, img_size, hipMemcpyHostToDevice);
	float block = 32;
	dim3 size_grid(ceil(columnas / block), ceil(filas / block), 1);
	dim3 size_block(block, block, 1);

	change_to_blur_kernel <<< size_grid, size_block >>> (d_img_in, d_img_out, columnas, filas);

	hipMemcpy(img_out, d_img_out, img_size, hipMemcpyDeviceToHost);
	hipFree(d_img_in);
	hipFree(d_img_out);
}

//CUDA code
__global__ void change_to_gray_kernel(OpenCVUnit* image, float* gray, unsigned int columnas, unsigned int filas) {
	unsigned int gpu_columma = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int gpu_fila = threadIdx.y + (blockIdx.y * blockDim.y);

	if (gpu_columma < columnas && gpu_fila < filas) {
		int place = gpu_fila * columnas + gpu_columma;
		gray[place] = (image[place].get_r() * 0.21 /*0.299*/) + (image[place].get_g() * 0.72/*0.587*/) + (image[place].get_b() * 0.07/*0.114*/);
	}
}
1

__global__ void change_to_blur_kernel(OpenCVUnit* image, OpenCVUnit* blur, unsigned int columnas, unsigned int filas) {
	unsigned int gpu_columna = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int gpu_fila = threadIdx.y + (blockIdx.y *  blockDim.y);

	if (gpu_columna < columnas && gpu_fila < filas) {
		int blur_size = 3;
		float in_r = 0.0;
		float in_g = 0.0;
		float in_b = 0.0;
		float size = 0;

		int in_fila, in_columna;
		for (int _fila = -blur_size; _fila < (blur_size + 1); _fila++) {
			for (int _col = blur_size; _col < blur_size + 1; _col++) {
				in_fila = gpu_fila + _fila;
				in_columna = gpu_columna + _col;
				if ( (in_fila > -1 && in_fila < filas) && (in_columna > -1 && in_columna < columnas) ) {
					in_r += image[(in_fila*columnas) + in_columna].get_r();
					in_g += image[(in_fila*columnas) + in_columna].get_g();
					in_b += image[(in_fila*columnas) + in_columna].get_b();
					size += 1;
				}
			}
		}

		blur[(gpu_fila*columnas) + gpu_columna].set_r(in_r / size);
		blur[(gpu_fila*columnas) + gpu_columna].set_g(in_g / size);
		blur[(gpu_fila*columnas) + gpu_columna].set_b(in_b / size);
	}
}


void print_matrix(OpenCVUnit* matriz, int fila, int columna) {
	for (int i = 0; i < fila; i++) {
		for (int j = 0; j < columna; j++) {
			cout << "[ " << matriz[i*columna + j].get_r() << " - " << matriz[i*columna + j].get_g() << " - " << matriz[i * columna + j].get_b() << " ]";
		}
	}
}

void print_matrix(float* matriz, int fila, int columna) {
	for (int i = 0; i < fila; i++) {
		for (int j = 0; j < columna; j++) {
			cout << "[ " << matriz[i * columna +  j] <<" ]";
		}
	}
}


void set_matrix_zero(float* matriz, int fila, int columna) {
	for (int i = 0; i < fila; i++) {
		for (int j = 0; j < columna; j++) {
			matriz[i*columna + j] = 0;
		}
	}
}